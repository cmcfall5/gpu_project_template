


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void make_hello(char *str, int *transform_mtx)
{
    str[threadIdx.x] += transform_mtx[threadIdx.x];
}

int main(int argc, char **argv)
{
    printf("Hello from main!\n");

    for (int ii = 0; ii < argc; ii++) {
        printf("argv[%d] = %s\n", ii, argv[ii]);
    }

    char str[16] = "Hello \0\0\0\0\0\0";
    int transform_mtx[16] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    printf("From CUDA: %s", str);
    char *ad;
    int *bd;
    hipMalloc((void**)&ad, sizeof(str)); 
    hipMalloc((void**)&bd, sizeof(transform_mtx)); 
    hipMemcpy(ad, str, sizeof(str), hipMemcpyHostToDevice); 
    hipMemcpy(bd, transform_mtx, sizeof(transform_mtx), hipMemcpyHostToDevice); 
    
    dim3 dimBlock(16, 1);
    dim3 dimGrid(1, 1);
    make_hello<<<dimGrid, dimBlock>>>(ad, bd);

    hipMemcpy(str, ad, sizeof(str), hipMemcpyDeviceToHost); 
    hipFree(ad);
    hipFree(bd);
    
    printf("%s\n", str);

    return 0;
}